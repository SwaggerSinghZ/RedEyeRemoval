#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include "../include/red_eye_removal.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void correctRedEyeKernel(unsigned char* data, int width, int height, int step, DetectionMethod method) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;

    int idx = y * step + x * 3;
    unsigned char B = data[idx];
    unsigned char G = data[idx + 1];
    unsigned char R = data[idx + 2];

    bool isRed = false;

    if (method == RGB_THRESHOLD) {
        isRed = (R > 150 && G < 100 && B < 100);
    } else if (method == NRR) {
        int sum = R + G + B;
        if (sum > 0)
            isRed = (float(R) / sum) > 0.6f;
    }

    if (isRed) {
        unsigned char avg = (G + B) / 2;
        data[idx] = data[idx + 1] = data[idx + 2] = avg;
    }
}

void detectAndCorrectRedEye_CUDA(const cv::Mat& input, DetectionMethod method, cv::Mat& output) {
    output = input.clone();

    cv::CascadeClassifier faceCascade, eyeCascade;
    faceCascade.load("haarcascade_frontalface_default.xml");
    eyeCascade.load("haarcascade_eye.xml");

    std::vector<cv::Rect> faces;
    faceCascade.detectMultiScale(input, faces);

    for (const auto& face : faces) {
        cv::Mat faceROI = input(face);
        std::vector<cv::Rect> eyes;
        eyeCascade.detectMultiScale(faceROI, eyes);

        for (const auto& eyeRect : eyes) {
            cv::Rect eyeAbs(eyeRect.x + face.x, eyeRect.y + face.y, eyeRect.width, eyeRect.height);
            cv::Mat eyeROI = output(eyeAbs);

            unsigned char* d_data;
            size_t step;
            hipMallocPitch(&d_data, &step, eyeROI.cols * 3, eyeROI.rows);
            hipMemcpy2D(d_data, step, eyeROI.data, eyeROI.step, eyeROI.cols * 3, eyeROI.rows, hipMemcpyHostToDevice);

            dim3 blockSize(16, 16);
            dim3 gridSize((eyeROI.cols + 15) / 16, (eyeROI.rows + 15) / 16);
            correctRedEyeKernel<<<gridSize, blockSize>>>(d_data, eyeROI.cols, eyeROI.rows, step, method);
            hipDeviceSynchronize();

            hipMemcpy2D(eyeROI.data, eyeROI.step, d_data, step, eyeROI.cols * 3, eyeROI.rows, hipMemcpyDeviceToHost);
            hipFree(d_data);
        }
    }
}
